#include <stdio.h>
#include <string.h>
#include <stdlib.h>
// CUDA Runtime
#include <hip/hip_runtime.h>
// Utility and system includes
#include <hip/hip_runtime_api.h>
// helper for shared that are common to CUDA Samples
#include <helper_functions.h>
#include <helper_timer.h>

#include "hist-equ.h"
#include "hist-equ.cuh"

void run_cpu_color_test(PPM_IMG img_in);
void run_gpu_color_test(PPM_IMG img_in);
void run_cpu_gray_test(PGM_IMG img_in);
void run_gpu_gray_test(PGM_IMG img_in);


int main(){
    PGM_IMG img_input_buffer_gray;
    PPM_IMG img_input_buffer_color;

    printf("Running contrast enhancement for gray-scale images.\n");
    img_input_buffer_gray = read_pgm("in.pgm");

    run_cpu_gray_test(img_input_buffer_gray);
    run_gpu_gray_test(img_input_buffer_gray);

    free_pgm(img_input_buffer_gray);



    printf("Running contrast enhancement for color images.\n");
    img_input_buffer_color = read_ppm("in.ppm");

    run_cpu_color_test(img_input_buffer_color);
    run_gpu_color_test(img_input_buffer_color);

    free_ppm(img_input_buffer_color);

    return 0;
}

void run_gpu_color_test(PPM_IMG img_in)
{
  StopWatchInterface *timer=NULL;
  printf("Starting GPU processing...\n");

  PPM_IMG yuv_image_output;

  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  // perform the enhancement
  yuv_image_output = gpu_contrast_enhancement_c_yuv(img_in);

  sdkStopTimer(&timer);
  printf("Processing time: %f (ms) for color yuv enhancement on GPU\n", sdkGetTimerValue(&timer));
  sdkDeleteTimer(&timer);

  write_ppm(yuv_image_output, "gpu_out_yuv.ppm");
  free_ppm(yuv_image_output);
}

void run_gpu_gray_test(PGM_IMG img_in)
{
    StopWatchInterface *timer=NULL;
    printf("Starting GPU processing...\n");

    PGM_IMG gray_image_output;

    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // perform the enhancement
    gray_image_output = gpu_contrast_enhancement_gray_image(img_in);

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms) for gray enhancement on GPU\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    write_pgm(gray_image_output, "gpu_out.pgm");
    free_pgm(gray_image_output);
}

void run_cpu_color_test(PPM_IMG img_in)
{
    StopWatchInterface *timer=NULL;
    PPM_IMG img_obuf_hsl, img_obuf_yuv;

    printf("Starting CPU processing...\n");

    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // perform the enhancement
    img_obuf_hsl = contrast_enhancement_c_hsl(img_in);

    sdkStopTimer(&timer);
    printf("HSL processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    write_ppm(img_obuf_hsl, "out_hsl.ppm");



    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // perform the enhancement
    img_obuf_yuv = contrast_enhancement_c_yuv(img_in);

    sdkStopTimer(&timer);
    printf("YUV processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    write_ppm(img_obuf_yuv, "out_yuv.ppm");


    free_ppm(img_obuf_hsl);
    free_ppm(img_obuf_yuv);
}




void run_cpu_gray_test(PGM_IMG img_in)
{
    StopWatchInterface *timer = NULL;
    PGM_IMG img_obuf;


    printf("Starting CPU processing...\n");

    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    // perform the enhancement
    img_obuf = contrast_enhancement_g(img_in);

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms) for gray enhancement on CPU\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    write_pgm(img_obuf, "out.pgm");
    free_pgm(img_obuf);
}



PPM_IMG read_ppm(const char * path){
    FILE * in_file;
    char sbuf[256];

    char *ibuf;
    PPM_IMG result;
    int v_max, i;
    in_file = fopen(path, "r");
    if (in_file == NULL){
        printf("Input file not found!\n");
        exit(1);
    }
    /*Skip the magic number*/
    fscanf(in_file, "%s", sbuf);
    fscanf(in_file, "%d",&result.w);
    fscanf(in_file, "%d",&result.h);
    fscanf(in_file, "%d\n",&v_max);

    printf("Image size: %d x %d\n", result.w, result.h);


    result.img_r = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    result.img_g = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    result.img_b = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    ibuf         = (char *)malloc(3 * result.w * result.h * sizeof(char));


    fread(ibuf,sizeof(unsigned char), 3 * result.w*result.h, in_file);

    for(i = 0; i < result.w*result.h; i ++){
        result.img_r[i] = ibuf[3*i + 0];
        result.img_g[i] = ibuf[3*i + 1];
        result.img_b[i] = ibuf[3*i + 2];
    }

    fclose(in_file);
    free(ibuf);

    return result;
}

void write_ppm(PPM_IMG img, const char * path){
    FILE * out_file;
    int i;

    char * obuf = (char *)malloc(3 * img.w * img.h * sizeof(char));

    for(i = 0; i < img.w*img.h; i ++){
        obuf[3*i + 0] = img.img_r[i];
        obuf[3*i + 1] = img.img_g[i];
        obuf[3*i + 2] = img.img_b[i];
    }
    out_file = fopen(path, "wb");
    fprintf(out_file, "P6\n");
    fprintf(out_file, "%d %d\n255\n",img.w, img.h);
    fwrite(obuf,sizeof(unsigned char), 3*img.w*img.h, out_file);
    fclose(out_file);
    free(obuf);
}

void free_ppm(PPM_IMG img)
{
    free(img.img_r);
    free(img.img_g);
    free(img.img_b);
}

PGM_IMG read_pgm(const char * path){
    FILE * in_file;
    char sbuf[256];


    PGM_IMG result;
    int v_max;//, i;
    in_file = fopen(path, "r");
    if (in_file == NULL){
        printf("Input file not found!\n");
        exit(1);
    }

    fscanf(in_file, "%s", sbuf); /*Skip the magic number*/
    fscanf(in_file, "%d",&result.w);
    fscanf(in_file, "%d",&result.h);
    fscanf(in_file, "%d\n",&v_max);
    printf("Image size: %d x %d\n", result.w, result.h);


    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));


    fread(result.img,sizeof(unsigned char), result.w*result.h, in_file);
    fclose(in_file);

    return result;
}

void write_pgm(PGM_IMG img, const char * path){
    FILE * out_file;
    out_file = fopen(path, "wb");
    fprintf(out_file, "P5\n");
    fprintf(out_file, "%d %d\n255\n",img.w, img.h);
    fwrite(img.img,sizeof(unsigned char), img.w*img.h, out_file);
    fclose(out_file);
}

void free_pgm(PGM_IMG img)
{
    free(img.img);
}
